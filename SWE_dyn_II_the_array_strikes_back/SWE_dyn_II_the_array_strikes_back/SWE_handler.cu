#include "hip/hip_runtime.h"
#include "SWE_handler.h"
using namespace std;

//constructor and destructor
SWE_handler::SWE_handler(int x, int y, float dx, float dy, float g, int rBX, int rBY, int mR, int bSX, int bSY)
{
	this->nx = x;
	this->ny = y;

	this->dx = dx;
	this->dy = dy;

	this->g = g;

	this->refinementBaseX = rBX;
	this->refinementBaseY = rBY;
	this->maxRecursions = mR;

	//important set synchronization level
	hipDeviceSetLimit(cudaLimitDevRuntimeSyncDepth, this->maxRecursions + 1);

	this->blockSize = dim3(bSX, bSY);

	//create array for adaptivity:
	tree = new ArrayHelper<int>(nx + 2, ny + 2);
	checkCudaErrors(hipMalloc(&d_tree, (nx + 2) * (ny + 2) * sizeof(int)));

	//create arrays for grids
	h = new ArrayHelper<float>(nx + 2, ny + 2);
	checkCudaErrors(hipMalloc(&d_h, (nx + 2) * (ny + 2) * sizeof(float)));
	hu = new ArrayHelper<float>(nx + 2, ny + 2);
	checkCudaErrors(hipMalloc(&d_hu, (nx + 2) * (ny + 2) * sizeof(float)));
	hv = new ArrayHelper<float>(nx + 2, ny + 2);
	checkCudaErrors(hipMalloc(&d_hv, (nx + 2) * (ny + 2) * sizeof(float)));
	b = new ArrayHelper<float>(nx + 2, ny + 2);
	checkCudaErrors(hipMalloc(&d_b, (nx + 2) * (ny + 2) * sizeof(float)));

	//and fluxes
	Bu = new ArrayHelper<float>(nx + 1, ny + 1);
	checkCudaErrors(hipMalloc(&d_Bu, (nx + 1) * (ny + 1) * sizeof(float)));
	Bv = new ArrayHelper<float>(nx + 1, ny + 1);
	checkCudaErrors(hipMalloc(&d_Bv, (nx + 1) * (ny + 1) * sizeof(float)));

	Fh = new ArrayHelper<float>(nx + 1, ny + 1);
	checkCudaErrors(hipMalloc(&d_Fh, (nx + 1) * (ny + 1) * sizeof(float)));
	Fhu = new ArrayHelper<float>(nx + 1, ny + 1);
	checkCudaErrors(hipMalloc(&d_Fhu, (nx + 1) * (ny + 1) * sizeof(float)));
	Fhv = new ArrayHelper<float>(nx + 1, ny + 1);
	checkCudaErrors(hipMalloc(&d_Fhv, (nx + 1) * (ny + 1) * sizeof(float)));
	Gh = new ArrayHelper<float>(nx + 1, ny + 1);
	checkCudaErrors(hipMalloc(&d_Gh, (nx + 1) * (ny + 1) * sizeof(float)));
	Ghu = new ArrayHelper<float>(nx + 1, ny + 1);
	checkCudaErrors(hipMalloc(&d_Ghu, (nx + 1) * (ny + 1) * sizeof(float)));
	Ghv = new ArrayHelper<float>(nx + 1, ny + 1);
	checkCudaErrors(hipMalloc(&d_Ghv, (nx + 1) * (ny + 1) * sizeof(float)));
}

SWE_handler::~SWE_handler()
{
	delete h;
	checkCudaErrors(hipFree(d_h));
	delete hu;
	checkCudaErrors(hipFree(d_hu));
	delete hv;
	checkCudaErrors(hipFree(d_hv));
	delete b;
	checkCudaErrors(hipFree(d_b));

	delete Bu;
	checkCudaErrors(hipFree(d_Bu));
	delete Bv;
	checkCudaErrors(hipFree(d_Bv));

	delete Fh;
	checkCudaErrors(hipFree(d_Fh));
	delete Fhu;
	checkCudaErrors(hipFree(d_Fhu));
	delete Fhv;
	checkCudaErrors(hipFree(d_Fhv));
	delete Gh;
	checkCudaErrors(hipFree(d_Gh));
	delete Ghu;
	checkCudaErrors(hipFree(d_Ghu));
	delete Ghv;
	checkCudaErrors(hipFree(d_Ghv));
}

//-------------------------------------------------
//initial values
void SWE_handler::setInitialValues(float h, float u, float v)
{
	//include border
	for (int i = 0; i <= nx + 1; i++)
	{
		for (int j = 0; j <= ny + 1; j++)
		{
			this->h->getValues()[computeIndex(this->h->getWidth(), this->h->getHeight(), i, j)] = h;
			this->hu->getValues()[computeIndex(this->hu->getWidth(), this->hu->getHeight(), i, j)] = h * u;
			this->hv->getValues()[computeIndex(this->hv->getWidth(), this->hv->getHeight(), i, j)] = h * v;

			//set the depths to maximum depth
			tree[computeIndex(this->h->getWidth(), this->h->getHeight(), i, j)] = this->maxRecursions;
		}
	}
	checkCudaErrors(hipMemcpy(this->d_h, this->h->getValues(), this->h->getWidth() * this->h->getHeight() * sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(this->d_hu, this->h->getValues(), this->h->getWidth() * this->h->getHeight() * sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(this->d_hv, this->h->getValues(), this->h->getWidth() * this->h->getHeight() * sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(this->d_tree, this->tree, this->h->getWidth() * this->h->getHeight() * sizeof(int), hipMemcpyHostToDevice));
}

void SWE_handler::setInitialValues(float(*h)(float, float), float u, float v)
{
	//include boundaries
	for (int i = 0; i <= nx + 1; i++)
	{
		for (int j = 0; j <= ny + 1; j++)
		{
			this->h->getValues()[computeIndex(this->h->getWidth(), this->h->getHeight(), i, j)] = h((i-0.5f)*dx, (j-0.5f)*dy);
			this->hu->getValues()[computeIndex(this->hu->getWidth(), this->hu->getHeight(), i, j)] = h((i-0.5f)*dx, (j-0.5f)*dy) * u;
			this->hv->getValues()[computeIndex(this->hv->getWidth(), this->hv->getHeight(), i, j)] = h((i-0.5f)*dx, (j-0.5f)*dy) * v;

			//set the depths to maximum depth
			tree[computeIndex(this->h->getWidth(), this->h->getHeight(), i, j)] = this->maxRecursions;
		}
	}
	checkCudaErrors(hipMemcpy(this->d_h, this->h->getValues(), this->h->getWidth() * this->h->getHeight() * sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(this->d_hu, this->h->getValues(), this->h->getWidth() * this->h->getHeight() * sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(this->d_hv, this->h->getValues(), this->h->getWidth() * this->h->getHeight() * sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(this->d_tree, this->tree, this->h->getWidth() * this->h->getHeight() * sizeof(int), hipMemcpyHostToDevice));
}

//-------------------------------------------------
//bathymetry values
void SWE_handler::setBathymetry(float b)
{
	//include border
	for (int i = 0; i < nx + 2; i++)
	{
		for (int j = 0; j < ny + 2; j++)
		{
			this->b->getValues()[computeIndex(this->b->getWidth(), this->b->getHeight(), i, j)] = b;
		}
	}
	checkCudaErrors(hipMemcpy(this->d_b, this->b->getValues(), this->b->getWidth() * this->b->getHeight() * sizeof(float), hipMemcpyHostToDevice));
	computeBathymetrySources();
}

void SWE_handler::setBathymetry(float(*b)(float, float))
{
	//include border
	for (int i = 0; i < nx + 2; i++)
	{
		for (int j = 0; j < ny + 2; j++)
		{
			this->b->getValues()[computeIndex(this->b->getWidth(), this->b->getHeight(), i, j)] = b((i - 0.5f)*dx, (j - 0.5f)*dy);
		}
	}
	checkCudaErrors(hipMemcpy(this->d_b, this->b->getValues(), this->b->getWidth() * this->b->getHeight() * sizeof(float), hipMemcpyHostToDevice));
	computeBathymetrySources();
}

void SWE_handler::computeBathymetrySources()
{
	//don't use tree refinement here, at least not yet
	dim3 blockDim = this->blockSize;
	dim3 gridDim(divUp(nx + 1, blockSize.x), divUp(ny + 1, blockSize.y));

	computeBathymetrySources_kernel << <gridDim, blockDim >> >(this->d_h, this->d_b, this->d_Bu, this->d_Bv, this->nx, this->ny this->g, this->maxRecursions);
	checkCudaErrors(hipGetLastError());
}

//-------------------------------------------------
//boundary computation
void SWE_handler::setBoundaryType(BoundaryType left, BoundaryType right, BoundaryType bottom, BoundaryType top)
{
	this->left = left;
	this->right = right;
	this->top = top;
	this->bottom = bottom;
}

void SWE_handler::setBoundaryLayer()
{
	//top bottom boundary
	dim3 horizontalBlock(this->blockSize.x * this->blockSize.x);
	dim3 horizontalGrid(divUp(this->h->getWidth(), horizontalBlock.x));

	setTopBorder_kernel << <horizontalGrid, horizontalBlock >> >(this->d_h, this->d_hu, this->d_hv, this->top);
	setBottomBorder_kernel << <horizontalGrid, horizontalBlock >> >(this->d_h, this->d_hu, this->d_hv, this->bottom);
	checkCudaErrors(hipGetLastError());

	//left right boundary
	dim3 verticalBlock(this->blockSize.y * this->blockSize.y);
	dim3 verticalGrid(divUp(this->h->getHeight(), horizontalBlock.x));

	setRightBorder_kernel << <verticalGrid, verticalBlock >> >(this->d_h, this->d_hu, this->d_hv, this->right);
	setLeftBorder_kernel << <verticalGrid, verticalBlock >> >(this->d_h, this->d_hu, this->d_hv, this->left);
	checkCudaErrors(hipGetLastError());
}

//-------------------------------------------------
//simulation
float SWE_handler::simulate(float startTime, float endTime)
{
	float t = startTime;

	do
	{
		setBoundaryLayer();

		computeBathymetrySources();

		t += eulerTimestep();
		cout << "currentTime: " << t << endl;

		//get max timestep for this
		//float tMax = getMaxTimestep();
		//this->setTimestep(tMax);

	} while (t < endTime);

	return t;
}

//-------------------------------------------------
//stepping forward in time
float SWE_handler::eulerTimestep()
{
	float pessimisticFactor = 0.5f;

	computeFluxes();

	//kernel using dynamic parallelism
	dim3 block = this->blockSize;
	dim3 grid = dim3(computeForestBase(this->nx, this->refinementBaseX, this->maxRecursions), computeForestBase(this->ny, this->refinementBaseX, this->maxRecursions));

	eulerTimestep_kernel << <grid, block >> >(this->d_h, this->d_hu, this->d_hv,
		this->d_Fh, this->d_Fhu, this->d_Fhv,
		this->d_Gh, this->d_Ghu, this->d_Ghv,
		this->d_Bu, this->d_Bv,
		this->dt, this->dx, this->dy,
		this->refinementBaseX, this->refinementBaseY, this->maxRecursions);

	return pessimisticFactor * dt;
}

//-------------------------------------------------
//fluxes
void SWE_handler::computeFluxes()
{
	dim3 blockDim = this->blockSize;
	dim3 gridDim(divUp(Fh->getWidth(), blockDim.x), divUp(Fh->getHeight(), blockDim.y));
	computeFluxesF_kernel << <gridDim, blockDim >> >(this->d_h, this->d_hu, this->d_hv, this->d_Fh, this->d_Fhu, this->d_Fhv, this->g);

	gridDim = dim3(divUp(Gh->getWidth(), blockDim.x), divUp(Gh->getHeight(), blockDim.y));
	computeFluxesG_kernel << <gridDim, blockDim >> >(this->d_h, this->d_hu, this->d_hv, this->d_Gh, this->d_Ghu, this->d_Ghv, this->g);
}

//-------------------------------------------------
//stepping forward in time
float SWE_handler::getMaxTimestep()
{
	float meshSize = (dx<dy) ? dx : dy;
	float hmax = 0.0f;
	float velmax = 0.0f;
	float2* result;

	dim3 block = this->blockSize;
	dim3 grid(computeForestBase(nx, refinementBaseX, maxRecursions), computeForestBase(ny, refinementBaseY, maxRecursions));

	hipMallocManaged(&result, grid.x * grid.y * sizeof(float2));

	//to be sure, that the simulation is finished
	checkCudaErrors(hipDeviceSynchronize());
	getMax_kernel << <grid, block >> >(this->d_h, this->d_hu, this->d_hv, result, grid.x, grid.y, this->refinementBaseX, this->refinementBaseY, this->maxRecursions);
	checkCudaErrors(hipDeviceSynchronize());

	for (unsigned int i = 0; i < grid.x * grid.y; i++)
	{
		hmax = max(hmax, result[i].x);
		velmax = max(velmax, result[i].y);
	}

	cout << "hmax: " << hmax << " velmax: " << velmax << endl;

	hipFree(result);

	return meshSize / (sqrtf(this->g * hmax) + velmax);
}

//-------------------------------------------------
//stepping forward in time
void SWE_handler::writeVTKFile(std::string filename)
{
	std::ofstream Vtk_file;
	// VTK HEADER
	Vtk_file.open(filename.c_str());
	Vtk_file << "# vtk DataFile Version 2.0" << endl;
	Vtk_file << "HPC Tutorials: Michael Bader, Kaveh Rahnema, Oliver Meister" << endl;
	Vtk_file << "ASCII" << endl;
	Vtk_file << "DATASET RECTILINEAR_GRID" << endl;
	Vtk_file << "DIMENSIONS " << nx + 1 << " " << ny + 1 << " " << "1" << endl;
	Vtk_file << "X_COORDINATES " << nx + 1 << " float" << endl;
	//GITTER PUNKTE
	for (int i = 0; i<nx + 1; i++)
		Vtk_file << i*dx << endl;
	Vtk_file << "Y_COORDINATES " << ny + 1 << " float" << endl;
	//GITTER PUNKTE
	for (int i = 0; i<ny + 1; i++)
		Vtk_file << i*dy << endl;
	Vtk_file << "Z_COORDINATES 1 float" << endl;
	Vtk_file << "0" << endl;
	Vtk_file << "CELL_DATA " << ny*nx << endl;
	Vtk_file << "SCALARS H float 1" << endl;
	Vtk_file << "LOOKUP_TABLE default" << endl;
	//DOFS
	for (int j = 1; j<ny + 1; j++)
		for (int i = 1; i<nx + 1; i++)
			Vtk_file << (h->getValues()[computeIndex(h->getWidth(), h->getHeight(), i, j)] + b->getValues()[computeIndex(b->getWidth(), b->getHeight(), i, j)]) << endl;
	Vtk_file << "SCALARS U float 1" << endl;
	Vtk_file << "LOOKUP_TABLE default" << endl;
	for (int j = 1; j<ny + 1; j++)
		for (int i = 1; i<nx + 1; i++)
			Vtk_file << hu->getValues()[computeIndex(hu->getWidth(), hu->getHeight(), i, j)] / h->getValues()[computeIndex(h->getWidth(), h->getHeight(), i, j)] << endl;
	Vtk_file << "SCALARS V float 1" << endl;
	Vtk_file << "LOOKUP_TABLE default" << endl;
	for (int j = 1; j<ny + 1; j++)
		for (int i = 1; i<nx + 1; i++)
			Vtk_file << hv->getValues()[computeIndex(hv->getWidth(), hv->getHeight(), i, j)] / h->getValues()[computeIndex(h->getWidth(), h->getHeight(), i, j)] << endl;
	Vtk_file << "SCALARS B float 1" << endl;
	Vtk_file << "LOOKUP_TABLE default" << endl;
	for (int j = 1; j<ny + 1; j++)
		for (int i = 1; i<nx + 1; i++)
			Vtk_file << b->getValues()[computeIndex(b->getWidth(), b->getHeight(), i, j)] << endl;
	Vtk_file.close();
}