#include "ArrayHelper.h"

ArrayHelper::ArrayHelper(int x, int y)
{
	nx = x;
	ny = y;

	hipMallocManaged(&values, nx * ny * sizeof(float));
	hipMallocManaged(&depths, nx * ny * sizeof(int));
}

TreeArray::~TreeArray()
{
	//crashes???
	//hipFree(values);
	//hipFree(depths);
}